// Comes from
// https://devtalk.nvidia.com/default/topic/1037482/gpu-accelerated-libraries/help-me-help-you-with-modern-cmake-and-cuda-mwe-for-npp/post/5271066/#5271066

#ifdef _WIN32
#  define EXPORT __declspec(dllexport)
#else
#  define EXPORT
#endif

#include <cstdio>

#include <assert.h>
#include <hip/hip_runtime_api.h>
#include <nppi_filtering_functions.h>

EXPORT int nppif_main()
{
  /**
   * 8-bit unsigned single-channel 1D row convolution.
   */
  int const simgrows = 32;
  int const simgcols = 32;
  Npp8u *d_pSrc, *d_pDst;
  int const nMaskSize = 3;
  NppiSize oROI;
  oROI.width = simgcols - nMaskSize;
  oROI.height = simgrows;
  int const simgsize = simgrows * simgcols * sizeof(d_pSrc[0]);
  int const dimgsize = oROI.width * oROI.height * sizeof(d_pSrc[0]);
  int const simgpix = simgrows * simgcols;
  int const dimgpix = oROI.width * oROI.height;
  int const nSrcStep = simgcols * sizeof(d_pSrc[0]);
  int const nDstStep = oROI.width * sizeof(d_pDst[0]);
  int const pixval = 1;
  int const nDivisor = 1;
  Npp32s const h_pKernel[nMaskSize] = { pixval, pixval, pixval };
  Npp32s* d_pKernel;
  Npp32s const nAnchor = 2;
  hipError_t err = hipMalloc((void**)&d_pSrc, simgsize);
  if (err != hipSuccess) {
    fprintf(stderr, "Cuda error %d\n", __LINE__);
    return 1;
  }
  err = hipMalloc((void**)&d_pDst, dimgsize);
  if (err != hipSuccess) {
    fprintf(stderr, "Cuda error %d\n", __LINE__);
    return 1;
  }
  err = hipMalloc((void**)&d_pKernel, nMaskSize * sizeof(d_pKernel[0]));
  if (err != hipSuccess) {
    fprintf(stderr, "Cuda error %d\n", __LINE__);
    return 1;
  }
  // set image to pixval initially
  err = hipMemset(d_pSrc, pixval, simgsize);
  if (err != hipSuccess) {
    fprintf(stderr, "Cuda error %d\n", __LINE__);
    return 1;
  }
  err = hipMemset(d_pDst, 0, dimgsize);
  if (err != hipSuccess) {
    fprintf(stderr, "Cuda error %d\n", __LINE__);
    return 1;
  }
  err = hipMemcpy(d_pKernel, h_pKernel, nMaskSize * sizeof(d_pKernel[0]),
                   hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr, "Cuda error %d\n", __LINE__);
    return 1;
  }
  // copy src to dst
  NppStatus ret =
    nppiFilterRow_8u_C1R(d_pSrc, nSrcStep, d_pDst, nDstStep, oROI, d_pKernel,
                         nMaskSize, nAnchor, nDivisor);
  assert(ret == NPP_NO_ERROR);
  Npp8u* h_imgres = new Npp8u[dimgpix];
  err = hipMemcpy(h_imgres, d_pDst, dimgsize, hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    fprintf(stderr, "Cuda error %d\n", __LINE__);
    return 1;
  }
  // test for filtering
  for (int i = 0; i < dimgpix; i++) {
    if (h_imgres[i] != (pixval * pixval * nMaskSize)) {
      fprintf(stderr, "h_imgres at index %d failed to match\n", i);
      return 1;
    }
  }

  return 0;
}
